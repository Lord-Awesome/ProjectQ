#include <hip/hip_complex.h>
#include <complex>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <stdexcept>
#include <fstream>
#include <chrono>

#define CUDACHECK(cmd) \
    hipError_t error=cmd; \
    if(error != hipSuccess) { \
        printf("%s in %s at line %d\n", hipGetErrorString(error),__ \
        FILE__,__LINE__); \
        exit(EXIT_FAILURE); \
    }

#define FILENAME "state_vec.txt"
#define MAT_FILENAME "source_matrix.txt"
#define MAT_DIM 8
#define C(r, i) make_hipComplex(r, i)
typedef hipComplex complex;

//Overload complex number functions
__device__ __host__ complex operator*(complex a, complex b) {return hipCmulf(a,b);}
__device__ __host__ complex operator+(complex a, complex b) {return hipCaddf(a,b);}
__device__ __host__ complex operator/(complex a, complex b) {return hipCdivf(a,b);}
__device__ __host__ bool operator==(complex a, complex b) {
	return a == b;
}
__device__ __host__ bool operator!=(complex a, complex b) {
	return a != b;
}
__host__ std::ostream & operator << (std::ostream &out, const complex &c) {
	out << "(" << hipCrealf(c);
	out << ",";
	out << hipCimagf(c) << ")\n";
	return out;
}
__host__ std::istream & operator >> (std::istream &in, complex &c) {
    char _; //throw out variable
    float real;
    float imag;
    in >> _; //"("
    in >> real;
    in >> _; //","
    in >> imag;
    in >> _ >> _; //")\n";
    c = C(real, imag);
    return in;
}

__constant__ complex operator_matrix[MAT_DIM][MAT_DIM];

std::chrono::high_resolution_clock::time_point start, stop;

__global__ void three_qubit_kernel(complex* vec, int vec_size, int qid0, int qid1, int qid2, int elements_per_chunk) {
    //qid0 is smaller than qid1

    //Initialize shared memory
    extern __shared__ complex smem[];

    int elements_per_thread = MAT_DIM; //3 quibit kernel

    int blocks_in_state_vector = ceil(vec_size / (float) (elements_per_thread * blockDim.x));
	int batch0_stride = 2 * (1 << qid0);
	int batch1_stride = 2 * (1 << qid1);
	int batch2_stride = 2 * (1 << qid2);
    for(int global_block_id = blockIdx.x; global_block_id < blocks_in_state_vector; global_block_id += gridDim.x) {

        int element_id_base = 0;
		int global_thread_id = (threadIdx.x + global_block_id * blockDim.x);

		int batch1_depth = global_thread_id % (batch1_stride/4);
		int batch0_id = batch1_depth / (batch0_stride/2);

		element_id_base += global_thread_id % (batch0_stride/2);
		element_id_base += batch0_id * batch0_stride;

		int batch2_depth = global_thread_id % (batch2_stride/8);
		int batch1_id = batch2_depth / (batch1_stride/4);

		element_id_base += batch1_id * batch1_stride;

		int batch2_id = global_thread_id / (batch2_stride/8);

		element_id_base += batch2_id * batch2_stride;


        //iteration dependent

        complex result[MAT_DIM];
        for(int row = 0; row < MAT_DIM; row++) {
            result[row] = C(0.0f, 0.0f);
        }
        for(int i = 0; i < 2; i++) {
            for(int j = 0; j < 2; j++) {
				for (int k = 0; k < 2; k++) {
					int offset = (i * (1 << qid2)) + (j * (1 << qid1)) + (k * (1 << qid0));
					int element_id = element_id_base + offset;

					//load
					complex val;
					if(element_id < vec_size) {
						val = vec[element_id];
					}
					else {
						val = C(0.0f,0.0f);
					}

					//compute
					int column = (4*i)+(2*j)+k;
					for(int row = 0; row < MAT_DIM; row++) {
						result[row] = result[row] + (operator_matrix[row][column]*val);
					}
				}//k
            }//j
        }//i

        for(int i = 0; i < 2; i++) {
            for(int j = 0; j < 2; j++) {
				for (int k = 0; k < 2; k++) {
					int offset = (i * (1 << qid2)) + (j * (1 << qid1)) + (k * (1 << qid0));
					int element_id = element_id_base + offset;

					//store
					int row = (4*i)+(2*j)+k;
					if(element_id < vec_size) {
						vec[element_id] = result[row];
						//vec[element_id] = C((float)element_id_base,(float)offset);
						//vec[element_id] = C((float)global_thread_id, (float)element_id_base);
					}
				}//k
            }//j
        }//i

    }
}

//TODO: Header
template <class M>
void run_kernel(complex* vec, int vec_size, int quid0, int quid1, int quid2, M source_matrix) {
    hipDeviceSynchronize();

    //Get smem size
    hipDeviceProp_t deviceProp;
    int dev_id = 0;
    hipGetDeviceProperties(&deviceProp, dev_id);
    int smem_size_in_bytes = (int) deviceProp.sharedMemPerBlock;
    int smem_size_in_elems = smem_size_in_bytes/(2*sizeof(double));

    int max_threads_per_block = (int) deviceProp.maxThreadsPerBlock;

	//A chunk can't be larger than shared memory because we need to hold it all at once
	//A chunk can't be larger than the threads in a block because we need one thread to handle each element
	//A chunk can't be larger than a batch by definition
    //int chunk_size = std::min(std::min(smem_size_in_elems, max_threads_per_block),(int) batch_size);
	int chunk_size = max_threads_per_block;
    int chunk_size_in_bytes = chunk_size * sizeof(complex);
    dim3 blockDim(chunk_size);
	int max_grid_size = deviceProp.maxGridSize[0];
    dim3 gridDim(std::min(max_grid_size, (int) ceil(vec_size/(float)chunk_size)));

    //print some stats about the GPU
    std::cout << "smem_size_in_elems: " << smem_size_in_elems << std::endl;
    std::cout << "max_threads_per_block: " << max_threads_per_block << std::endl;;
    std::cout << "chunk size: " << chunk_size << std::endl;;
    std::cout << "max grid size: " << max_grid_size << std::endl;;

	std::cout << "Vec size (num vectors is log2): " << vec_size << std::endl;
	std::cout << "quid0: " << quid0 << std::endl;
	std::cout << "quid1: " << quid1 << std::endl;
	std::cout << "quid2: " << quid2 << std::endl;

    std::cout << "block dim: " << blockDim.x << std::endl;
    std::cout << "grid dim: " << gridDim.x << std::endl;

    //memcpy and run the kernel
	start = std::chrono::high_resolution_clock::now();

    complex *d_vec;
    hipMalloc((void **) &d_vec, vec_size*sizeof(complex));
    hipError_t cpy_error = hipMemcpy(d_vec, vec, vec_size*sizeof(complex), hipMemcpyHostToDevice);
	//std::cout << "Copying to device error is: " << cpy_error << std::endl;
    three_qubit_kernel<<<gridDim, blockDim, chunk_size_in_bytes>>>(d_vec, vec_size, quid0, quid1, quid2, chunk_size);
	//cudaError_t kernel_error = cudaGetLastError();
	//std::cout << "Kernel error is: " << kernel_error << std::endl;
    hipDeviceSynchronize();
    cpy_error = hipMemcpy(vec, d_vec, vec_size*sizeof(complex), hipMemcpyDeviceToHost);
	//std::cout << "Copying to host error is: " << cpy_error << std::endl;
    hipFree(d_vec);

	stop = std::chrono::high_resolution_clock::now();
}

int main(int argc, char **argv) {

	if (argc != 5) {
		std::cout << "Input args wrong. Needs exactly four input args" << std::endl;
		exit(1);
	}


	int quid0 = atoi(argv[2]);
	int quid1 = atoi(argv[3]);
	int quid2 = atoi(argv[4]);

    //Read state vector
    std::vector<complex> state_vec;
    std::ifstream fin;
    complex temp;
	std::complex<float> std_complex_temp;


    fin.open(FILENAME);
    while(fin >> std_complex_temp) {
		temp = C(std_complex_temp.real(), std_complex_temp.imag());
        state_vec.push_back(temp);
    }
	if (fin.rdstate() == std::ios_base::failbit) {
		std::cout << "Ifstream failed with failbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::eofbit) {
		std::cout << "Ifstream failed with eofbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::badbit) {
		std::cout << "Ifstream failed with badbit" << std::endl;
	}
	std::cout << "Vector size: " << state_vec.size() << std::endl;
    fin.close();

	/*
    for (unsigned long i = 0; i < 1 << atoi(argv[1]); i++){ 
        //Note: normalization ignored for now
        float real = ((float) rand() / (float) (RAND_MAX));
        float imag = ((float) rand() / (float) (RAND_MAX));
        complex val = C(real, imag);
		state_vec.push_back(val);
    }
	*/

    unsigned long state_vec_size = state_vec.size();


    std::vector<complex> source_matrix_vec;
	std::cout << "here is the source matrix: " << std::endl;
    fin.open(MAT_FILENAME);
    while(fin >> std_complex_temp) {
		temp = C(std_complex_temp.real(), std_complex_temp.imag());
        source_matrix_vec.push_back(temp);
		std::cout << temp << std::endl;
    }
	if (fin.rdstate() == std::ios_base::failbit) {
		std::cout << "Ifstream failed with failbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::eofbit) {
		std::cout << "Ifstream failed with eofbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::badbit) {
		std::cout << "Ifstream failed with badbit" << std::endl;
	}
    fin.close();

    //Apply gate
    //Fill operator matrix in const mem
    hipMemcpyToSymbol(HIP_SYMBOL(operator_matrix), source_matrix_vec.data(), MAT_DIM * MAT_DIM * sizeof(complex), 0, hipMemcpyHostToDevice);

    run_kernel(state_vec.data(), state_vec_size, quid0, quid1, quid2, source_matrix_vec.data());


	auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
	std::cout << "GPU kernel execution time: " << duration.count() << std::endl;

	std::ofstream f_time;
	f_time.open("time_comparison.txt", std::ios_base::app);
	f_time << "GPU time: " << duration.count() << "\n";
	f_time.close();
 
    std::ofstream f;
    f.open("output.txt");
    for (unsigned long i = 0; i < state_vec_size; ++i) {
        complex val = state_vec[i];
        f << val;	
    }
    f.close();
    
    //debug
    std::cout << "size: " << state_vec.size() << std::endl;
    std::cout << state_vec.back() << std::endl;


    return 0;
}
