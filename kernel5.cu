#include <hip/hip_complex.h>
#include <complex>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <stdexcept>
#include <fstream>
#include <chrono>

#define CUDACHECK(cmd) \
    hipError_t error=cmd; \
    if(error != hipSuccess) { \
        printf("%s in %s at line %d\n", hipGetErrorString(error),__ \
        FILE__,__LINE__); \
        exit(EXIT_FAILURE); \
    }

#define FILENAME "state_vec.txt"
#define MAT_FILENAME "source_matrix.txt"
#define MAT_DIM 32
#define C(r, i) make_hipComplex(r, i)
typedef hipComplex complex;

//Overload complex number functions
__device__ __host__ complex operator*(complex a, complex b) {return hipCmulf(a,b);}
__device__ __host__ complex operator+(complex a, complex b) {return hipCaddf(a,b);}
__device__ __host__ complex operator/(complex a, complex b) {return hipCdivf(a,b);}
__device__ __host__ bool operator==(complex a, complex b) {
	return a == b;
}
__device__ __host__ bool operator!=(complex a, complex b) {
	return a != b;
}
__host__ std::ostream & operator << (std::ostream &out, const complex &c) {
	out << "(" << hipCrealf(c);
	out << ",";
	out << hipCimagf(c) << ")\n";
	return out;
}
__host__ std::istream & operator >> (std::istream &in, complex &c) {
    char _; //throw out variable
    float real;
    float imag;
    in >> _; //"("
    in >> real;
    in >> _; //","
    in >> imag;
    in >> _ >> _; //")\n";
    c = C(real, imag);
    return in;
}

__constant__ complex operator_matrix[MAT_DIM][MAT_DIM];

std::chrono::high_resolution_clock::time_point start, stop;

__global__ void five_qubit_kernel(complex* vec, int vec_size, int qid0, int qid1, int qid2, int qid3, int qid4, int elements_per_chunk) {
    //qid0 is smaller than qid1

    //Initialize shared memory
    __shared__ int smem[20];

    int blocks_in_state_vector = ceil(vec_size / (float) (MAT_DIM * blockDim.x));
	//int flag = 0;
    for(int global_block_id = blockIdx.x; global_block_id < blocks_in_state_vector; global_block_id += gridDim.x) {
		__syncthreads();

        //inside batch0
        //int blocks_per_batch0 = (1 << qid0) / elements_per_chunk;
        smem[0] = (1 << qid0) / elements_per_chunk;
        //int batch0_stride = 2 * (1 << qid0);
        smem[1] = 2 * (1 << qid0);

        //inside batch1
        //int blocks_per_batch1 = (1 << qid1) / (2 * elements_per_chunk);
        smem[2] = (1 << qid1) / (2 * elements_per_chunk);
        //int batch1_depth = (global_block_id % smem[2]);
        smem[3] = (global_block_id % smem[2]);
        //int batch1_stride = 2 * (1 << qid1);
        smem[4] = 2 * (1 << qid1);

		//inside batch2
		//int blocks_per_batch2 = (1 << qid2) / (4 * elements_per_chunk);
		smem[5] = (1 << qid2) / (4 * elements_per_chunk);
		//int batch2_depth = (global_block_id % smem[5]);
		smem[6] = (global_block_id % smem[5]);
		//int batch2_stride = 2 * (1 << qid2);
		smem[7] = 2 * (1 << qid2);

		//inside batch3
		//int blocks_per_batch3 = (1 << qid3) / (8 * elements_per_chunk);
		smem[8] = (1 << qid3) / (8 * elements_per_chunk);
		//int batch3_depth = (global_block_id % smem[8]);
		smem[9] = (global_block_id % smem[8]);
		//int batch3_stride = 2 * (1 << qid3);
		smem[10] = 2 * (1 << qid3);

		//inside batch4
		//int blocks_per_batch4 = (1 << qid4) / (16 * elements_per_chunk);
		smem[11] = (1 << qid4) / (16 * elements_per_chunk);
		//int batch4_depth = (global_block_id % smem[11]);
		smem[12] = (global_block_id % smem[11]);
		//int batch4_stride = 2 * (1 << qid4);
		smem[13] = 2 * (1 << qid4);

        //ids
        //int chunk_id = global_block_id % smem[0];
        smem[14] = global_block_id % smem[0];
        //int batch0_id = smem[3] / smem[0];
        smem[15] = smem[3] / smem[0];
        //int batch1_id = smem[6] / smem[2];
        smem[16] = smem[6] / smem[2];
        //int batch2_id = smem[9] / smem[5];
        smem[17] = smem[9] / smem[5];
        //int batch3_id = smem[12] / smem[8];
        smem[18] = smem[12] / smem[8];
        //int batch4_id = global_block_id / smem[11];
        smem[19] = global_block_id / smem[11];

        int element_id_base = 0;
        element_id_base += threadIdx.x;
        element_id_base += smem[14] * elements_per_chunk;
        element_id_base += smem[15] * smem[1];
        element_id_base += smem[16] * smem[4];
        element_id_base += smem[17] * smem[7];
        element_id_base += smem[18] * smem[10];
        element_id_base += smem[19] * smem[13];

        //iteration dependent

#define compute(i,j,k,l,m)\
		{\
		int offset = (i * (1 << qid4)) + (j * (1 << qid3)) + (k * (1 << qid2)) + (l * (1 << qid1)) + (m * (1 << qid0));\
		int element_id = element_id_base + offset;\
		complex val = C(0.0f,0.0f);\
		if(element_id < vec_size) val = vec[element_id];\
		int column = (16*i)+(8*j)+(4*k)+(2*l)+m;\
		for(int row = 0; row < MAT_DIM; row++) result[row] = result[row] + (operator_matrix[row][column]*val);\
		}


        __shared__ complex result[MAT_DIM];
        for(int row = 0; row < MAT_DIM; row++) result[row] = C(0.0f, 0.0f);


		compute(0,0,0,0,0);
		compute(0,0,0,0,1);
		compute(0,0,0,1,0);
		compute(0,0,0,1,1);
		compute(0,0,1,0,0);
		compute(0,0,1,0,1);
		compute(0,0,1,1,0);
		compute(0,0,1,1,1);
		compute(0,1,0,0,0);
		compute(0,1,0,0,1);
		compute(0,1,0,1,0);
		compute(0,1,0,1,1);
		compute(0,1,1,0,0);
		compute(0,1,1,0,1);
		compute(0,1,1,1,0);
		compute(0,1,1,1,1);
		compute(1,0,0,0,0);
		compute(1,0,0,0,1);
		compute(1,0,0,1,0);
		compute(1,0,0,1,1);
		compute(1,0,1,0,0);
		compute(1,0,1,0,1);
		compute(1,0,1,1,0);
		compute(1,0,1,1,1);
		compute(1,1,0,0,0);
		compute(1,1,0,0,1);
		compute(1,1,0,1,0);
		compute(1,1,0,1,1);
		compute(1,1,1,0,0);
		compute(1,1,1,0,1);
		compute(1,1,1,1,0);
		compute(1,1,1,1,1);


		#define store(i,j,k,l,m) \
		{\
		int offset = (i * (1 << qid4)) + (j * (1 << qid3)) + (k * (1 << qid2)) + (l * (1 << qid1)) + (m * (1 << qid0));\
		int element_id = element_id_base + offset;\
		int row = (16*i)+(8*j)+(4*k)+(2*l)+m;\
		if(element_id < vec_size) vec[element_id] = result[row];\
		}

		store(0,0,0,0,0);
		store(0,0,0,0,1);
		store(0,0,0,1,0);
		store(0,0,0,1,1);
		store(0,0,1,0,0);
		store(0,0,1,0,1);
		store(0,0,1,1,0);
		store(0,0,1,1,1);
		store(0,1,0,0,0);
		store(0,1,0,0,1);
		store(0,1,0,1,0);
		store(0,1,0,1,1);
		store(0,1,1,0,0);
		store(0,1,1,0,1);
		store(0,1,1,1,0);
		store(0,1,1,1,1);
		store(1,0,0,0,0);
		store(1,0,0,0,1);
		store(1,0,0,1,0);
		store(1,0,0,1,1);
		store(1,0,1,0,0);
		store(1,0,1,0,1);
		store(1,0,1,1,0);
		store(1,0,1,1,1);
		store(1,1,0,0,0);
		store(1,1,0,0,1);
		store(1,1,0,1,0);
		store(1,1,0,1,1);
		store(1,1,1,0,0);
		store(1,1,1,0,1);
		store(1,1,1,1,0);
		store(1,1,1,1,1);

    }
}

//TODO: Header
template <class M>
void run_kernel(complex* vec, int vec_size, int quid0, int quid1, int quid2, int quid3, int quid4, M source_matrix) {
    hipDeviceSynchronize();

    //Get smem size
    hipDeviceProp_t deviceProp;
    int dev_id = 0;
    hipGetDeviceProperties(&deviceProp, dev_id);
    int smem_size_in_bytes = (int) deviceProp.sharedMemPerBlock;
    int smem_size_in_elems = smem_size_in_bytes/(2*sizeof(double));

    int max_threads_per_block = (int) deviceProp.maxThreadsPerBlock;

    //batch: pairs before regions overlap
	const unsigned long batch_size = 1UL << (quid0); //in elements

	//A chunk can't be larger than shared memory because we need to hold it all at once
	//A chunk can't be larger than the threads in a block because we need one thread to handle each element
	//A chunk can't be larger than a batch by definition
    int chunk_size = std::min(std::min(smem_size_in_elems, max_threads_per_block),(int) batch_size);
	//CRK: Full blocks are allocating too many resources (not enough registers per block), so I'm nerfing it
	if (chunk_size > 512) {
		chunk_size = 512;
	}
    int chunk_size_in_bytes = chunk_size * sizeof(complex);
    dim3 blockDim(chunk_size);
	int max_grid_size = deviceProp.maxGridSize[0];
    dim3 gridDim(std::min(max_grid_size, (int) ceil(vec_size/(float)chunk_size)));

	size_t max_const_mem = deviceProp.totalConstMem;

	int max_reg_per_block = deviceProp.regsPerBlock;

    //print some stats about the GPU
    std::cout << "smem_size_in_elems: " << smem_size_in_elems << std::endl;
    std::cout << "max_threads_per_block: " << max_threads_per_block << std::endl;;
    std::cout << "batch size: " << batch_size << std::endl;;
    std::cout << "chunk size: " << chunk_size << std::endl;;
    std::cout << "max grid size: " << max_grid_size << std::endl;;
	std::cout << "const mem on device: " << max_const_mem << std::endl;
	std::cout << "max registers per block: " << max_reg_per_block << std::endl;

	std::cout << "Vec size (num vectors is log2): " << vec_size << std::endl;
	std::cout << "quid0: " << quid0 << std::endl;
	std::cout << "quid1: " << quid1 << std::endl;
	std::cout << "quid2: " << quid2 << std::endl;
	std::cout << "quid3: " << quid3 << std::endl;
	std::cout << "quid4: " << quid4 << std::endl;

    std::cout << "block dim: " << blockDim.x << std::endl;
    std::cout << "grid dim: " << gridDim.x << std::endl;

    //memcpy and run the kernel
	start = std::chrono::high_resolution_clock::now();

    complex *d_vec;
    hipError_t malloc_error = hipMalloc((void **) &d_vec, vec_size*sizeof(complex));
	std::cout << "Malloc error is: " << malloc_error << std::endl;
    hipError_t cpy_error = hipMemcpy(d_vec, vec, vec_size*sizeof(complex), hipMemcpyHostToDevice);
	std::cout << "Copying to device error is: " << cpy_error << std::endl;
    five_qubit_kernel<<<gridDim, blockDim, chunk_size_in_bytes>>>(d_vec, vec_size, quid0, quid1, quid2, quid3, quid4, chunk_size);
    hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel failed with error: " << hipGetErrorString(err) << std::endl;
	}
    cpy_error = hipMemcpy(vec, d_vec, vec_size*sizeof(complex), hipMemcpyDeviceToHost);
	std::cout << "Copying to host error is: " << cpy_error << std::endl;
    hipFree(d_vec);

	stop = std::chrono::high_resolution_clock::now();
}

int main(int argc, char **argv) {

	if (argc != 7) {
		std::cout << "Input args wrong. Needs exactly six input args" << std::endl;
		exit(1);
	}


	int quid0 = atoi(argv[2]);
	int quid1 = atoi(argv[3]);
	int quid2 = atoi(argv[4]);
	int quid3 = atoi(argv[5]);
	int quid4 = atoi(argv[6]);

    //Read state vector
    std::vector<complex> state_vec;
    std::ifstream fin;
    complex temp;
	std::complex<float> std_complex_temp;
	/*
    fin.open(FILENAME);
    while(fin >> std_complex_temp) {
		temp = C(std_complex_temp.real(), std_complex_temp.imag());
        state_vec.push_back(temp);
    }
	if (fin.rdstate() == std::ios_base::failbit) {
		std::cout << "Ifstream failed with failbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::eofbit) {
		std::cout << "Ifstream failed with eofbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::badbit) {
		std::cout << "Ifstream failed with badbit" << std::endl;
	}
	std::cout << "Vector size: " << state_vec_size << std::endl;
    fin.close();
	*/
    for (unsigned long i = 0; i < 1 << atoi(argv[1]); i++){ 
        //Note: normalization ignored for now
        float real = ((float) rand() / (float) (RAND_MAX));
        float imag = ((float) rand() / (float) (RAND_MAX));
        complex val = C(real, imag);
		state_vec.push_back(val);
    }

    unsigned long state_vec_size = state_vec.size();


    std::vector<complex> source_matrix_vec;
	std::cout << "here is the source matrix: " << std::endl;
    fin.open(MAT_FILENAME);
    while(fin >> std_complex_temp) {
		temp = C(std_complex_temp.real(), std_complex_temp.imag());
		std::cout << temp << std::endl;
		source_matrix_vec.push_back(temp);
    }
	if (fin.rdstate() == std::ios_base::failbit) {
		std::cout << "Ifstream failed with failbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::eofbit) {
		std::cout << "Ifstream failed with eofbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::badbit) {
		std::cout << "Ifstream failed with badbit" << std::endl;
	}
    fin.close();

    //Apply gate
    //Fill operator matrix in const mem
    hipError_t symbol_error = hipMemcpyToSymbol(HIP_SYMBOL(operator_matrix), source_matrix_vec.data(), MAT_DIM * MAT_DIM * sizeof(complex), 0, hipMemcpyHostToDevice);
	std::cout << "Error from symbol copy is: " << symbol_error << std::endl;

    run_kernel(state_vec.data(), state_vec_size, quid0, quid1, quid2, quid3, quid4, source_matrix_vec.data());


	auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
	std::cout << "GPU kernel execution time: " << duration.count() << std::endl;

	std::ofstream f_time;
	f_time.open("time_comparison.txt", std::ios_base::app);
	f_time << "GPU time: " << duration.count() << "\n";
	f_time.close();
 
    std::ofstream f;
    f.open("output.txt");
    for (unsigned long i = 0; i < state_vec_size; ++i) {
        complex val = state_vec[i];
        f << val;	
    }
    f.close();
    
    //debug
    std::cout << "size: " << state_vec.size() << std::endl;
    std::cout << state_vec.back() << std::endl;


    return 0;
}
