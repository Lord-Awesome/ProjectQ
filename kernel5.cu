#include <hip/hip_complex.h>
#include <complex>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <stdexcept>
#include <fstream>
#include <chrono>

#define CUDACHECK(cmd) \
    hipError_t error=cmd; \
    if(error != hipSuccess) { \
        printf("%s in %s at line %d\n", hipGetErrorString(error),__ \
        FILE__,__LINE__); \
        exit(EXIT_FAILURE); \
    }

#define FILENAME "state_vec.txt"
#define MAT_FILENAME "source_matrix.txt"
#define MAT_DIM 32
#define C(r, i) make_hipComplex(r, i)
typedef hipComplex complex;

//Overload complex number functions
__device__ __host__ complex operator*(complex a, complex b) {return hipCmulf(a,b);}
__device__ __host__ complex operator+(complex a, complex b) {return hipCaddf(a,b);}
__device__ __host__ complex operator/(complex a, complex b) {return hipCdivf(a,b);}
__device__ __host__ bool operator==(complex a, complex b) {
	return a == b;
}
__device__ __host__ bool operator!=(complex a, complex b) {
	return a != b;
}
__host__ std::ostream & operator << (std::ostream &out, const complex &c) {
	out << "(" << hipCrealf(c);
	out << ",";
	out << hipCimagf(c) << ")\n";
	return out;
}
__host__ std::istream & operator >> (std::istream &in, complex &c) {
    char _; //throw out variable
    float real;
    float imag;
    in >> _; //"("
    in >> real;
    in >> _; //","
    in >> imag;
    in >> _ >> _; //")\n";
    c = C(real, imag);
    return in;
}

__constant__ complex operator_matrix[MAT_DIM][MAT_DIM];

std::chrono::high_resolution_clock::time_point start, stop;

__global__ void five_qubit_kernel(complex* vec, int vec_size, int qid0, int qid1, int qid2, int qid3, int qid4, int elements_per_chunk) {
    //qid0 is smaller than qid1

    //Initialize shared memory
    __shared__ int smem[20];

    int blocks_in_state_vector = ceil(vec_size / (float) (MAT_DIM * blockDim.x));
    for(int global_block_id = blockIdx.x; global_block_id < blocks_in_state_vector; global_block_id += gridDim.x) {
		__syncthreads();

		int batch0_stride = 2 * (1 << qid0);
		int batch1_stride = 2 * (1 << qid1);
		int batch2_stride = 2 * (1 << qid2);
		int batch3_stride = 2 * (1 << qid3);
		int batch4_stride = 2 * (1 << qid4);

        int element_id_base = 0;
		int global_thread_id = (threadIdx.x + global_block_id * blockDim.x);
		{
		element_id_base += global_thread_id % (batch0_stride/2);
		}

		{
		int batch1_depth = global_thread_id % (batch1_stride/4);
		int batch0_id = batch1_depth / (batch0_stride/2);

		element_id_base += batch0_id * batch0_stride;
		}

		{
		int batch2_depth = global_thread_id % (batch2_stride/8);
		int batch1_id = batch2_depth / (batch1_stride/4);

		element_id_base += batch1_id * batch1_stride;
		}

		{
		int batch3_depth = global_thread_id % (batch3_stride/16);
		int batch2_id = batch3_depth / (batch2_stride/8);

		element_id_base += batch2_id * batch2_stride;
		}

		{
		int batch4_depth = global_thread_id % (batch4_stride/32);
		int batch3_id = batch4_depth / (batch3_stride/16);

		element_id_base += batch3_id * batch3_stride;
		}

		{
		int batch4_id = global_thread_id / (batch4_stride/32);

		element_id_base += batch4_id * batch4_stride;
		}

        //iteration dependent

#define compute(i,j,k,l,m)\
		{\
		int offset = (i * (1 << qid4)) + (j * (1 << qid3)) + (k * (1 << qid2)) + (l * (1 << qid1)) + (m * (1 << qid0));\
		int element_id = element_id_base + offset;\
		complex val = C(0.0f,0.0f);\
		if(element_id < vec_size) val = vec[element_id];\
		int column = (16*i)+(8*j)+(4*k)+(2*l)+m;\
		for(int row = 0; row < MAT_DIM; row++) result[row] = result[row] + (operator_matrix[row][column]*val);\
		}


		complex result[MAT_DIM];
        for(int row = 0; row < MAT_DIM; row++) result[row] = C(0.0f, 0.0f);


		compute(0,0,0,0,0);
		compute(0,0,0,0,1);
		compute(0,0,0,1,0);
		compute(0,0,0,1,1);
		compute(0,0,1,0,0);
		compute(0,0,1,0,1);
		compute(0,0,1,1,0);
		compute(0,0,1,1,1);
		compute(0,1,0,0,0);
		compute(0,1,0,0,1);
		compute(0,1,0,1,0);
		compute(0,1,0,1,1);
		compute(0,1,1,0,0);
		compute(0,1,1,0,1);
		compute(0,1,1,1,0);
		compute(0,1,1,1,1);
		compute(1,0,0,0,0);
		compute(1,0,0,0,1);
		compute(1,0,0,1,0);
		compute(1,0,0,1,1);
		compute(1,0,1,0,0);
		compute(1,0,1,0,1);
		compute(1,0,1,1,0);
		compute(1,0,1,1,1);
		compute(1,1,0,0,0);
		compute(1,1,0,0,1);
		compute(1,1,0,1,0);
		compute(1,1,0,1,1);
		compute(1,1,1,0,0);
		compute(1,1,1,0,1);
		compute(1,1,1,1,0);
		compute(1,1,1,1,1);


		#define store(i,j,k,l,m) \
		{\
		int offset = (i * (1 << qid4)) + (j * (1 << qid3)) + (k * (1 << qid2)) + (l * (1 << qid1)) + (m * (1 << qid0));\
		int element_id = element_id_base + offset;\
		int row = (16*i)+(8*j)+(4*k)+(2*l)+m;\
		if(element_id < vec_size) vec[element_id] = result[row];\
		}

		store(0,0,0,0,0);
		store(0,0,0,0,1);
		store(0,0,0,1,0);
		store(0,0,0,1,1);
		store(0,0,1,0,0);
		store(0,0,1,0,1);
		store(0,0,1,1,0);
		store(0,0,1,1,1);
		store(0,1,0,0,0);
		store(0,1,0,0,1);
		store(0,1,0,1,0);
		store(0,1,0,1,1);
		store(0,1,1,0,0);
		store(0,1,1,0,1);
		store(0,1,1,1,0);
		store(0,1,1,1,1);
		store(1,0,0,0,0);
		store(1,0,0,0,1);
		store(1,0,0,1,0);
		store(1,0,0,1,1);
		store(1,0,1,0,0);
		store(1,0,1,0,1);
		store(1,0,1,1,0);
		store(1,0,1,1,1);
		store(1,1,0,0,0);
		store(1,1,0,0,1);
		store(1,1,0,1,0);
		store(1,1,0,1,1);
		store(1,1,1,0,0);
		store(1,1,1,0,1);
		store(1,1,1,1,0);
		store(1,1,1,1,1);

    }
}

//TODO: Header
template <class M>
void run_kernel(complex* vec, int vec_size, int quid0, int quid1, int quid2, int quid3, int quid4, M source_matrix) {
    hipDeviceSynchronize();

    //Get smem size
    hipDeviceProp_t deviceProp;
    int dev_id = 0;
    hipGetDeviceProperties(&deviceProp, dev_id);
    int smem_size_in_bytes = (int) deviceProp.sharedMemPerBlock;
    int smem_size_in_elems = smem_size_in_bytes/(2*sizeof(double));

    int max_threads_per_block = (int) deviceProp.maxThreadsPerBlock;

    //batch: pairs before regions overlap
	const unsigned long batch_size = 1UL << (quid0); //in elements

	//A chunk can't be larger than shared memory because we need to hold it all at once
	//A chunk can't be larger than the threads in a block because we need one thread to handle each element
	//A chunk can't be larger than a batch by definition
    //int chunk_size = std::min(std::min(smem_size_in_elems, max_threads_per_block),(int) batch_size);
	int chunk_size = 256;
    int chunk_size_in_bytes = chunk_size * sizeof(complex);
    dim3 blockDim(chunk_size);
	int max_grid_size = deviceProp.maxGridSize[0];
    dim3 gridDim(std::min(max_grid_size, (int) ceil(vec_size/(float)chunk_size)));

	size_t max_const_mem = deviceProp.totalConstMem;

	int max_reg_per_block = deviceProp.regsPerBlock;

    //print some stats about the GPU
    std::cout << "smem_size_in_elems: " << smem_size_in_elems << std::endl;
    std::cout << "max_threads_per_block: " << max_threads_per_block << std::endl;;
    std::cout << "batch size: " << batch_size << std::endl;;
    std::cout << "chunk size: " << chunk_size << std::endl;;
    std::cout << "max grid size: " << max_grid_size << std::endl;;
	std::cout << "const mem on device: " << max_const_mem << std::endl;
	std::cout << "max registers per block: " << max_reg_per_block << std::endl;

	std::cout << "Vec size (num vectors is log2): " << vec_size << std::endl;
	std::cout << "quid0: " << quid0 << std::endl;
	std::cout << "quid1: " << quid1 << std::endl;
	std::cout << "quid2: " << quid2 << std::endl;
	std::cout << "quid3: " << quid3 << std::endl;
	std::cout << "quid4: " << quid4 << std::endl;

    std::cout << "block dim: " << blockDim.x << std::endl;
    std::cout << "grid dim: " << gridDim.x << std::endl;

    //memcpy and run the kernel

    complex *d_vec;
    hipError_t malloc_error = hipMalloc((void **) &d_vec, vec_size*sizeof(complex));
	//std::cout << "Malloc error is: " << malloc_error << std::endl;
    hipError_t cpy_error = hipMemcpy(d_vec, vec, vec_size*sizeof(complex), hipMemcpyHostToDevice);
	//std::cout << "Copying to device error is: " << cpy_error << std::endl;
	start = std::chrono::high_resolution_clock::now();
    hipMemcpyToSymbol(HIP_SYMBOL(operator_matrix), source_matrix, MAT_DIM * MAT_DIM * sizeof(complex), 0, hipMemcpyHostToDevice);
    five_qubit_kernel<<<gridDim, blockDim, chunk_size_in_bytes>>>(d_vec, vec_size, quid0, quid1, quid2, quid3, quid4, chunk_size);
	stop = std::chrono::high_resolution_clock::now();
    hipDeviceSynchronize();
	//cudaError_t err = cudaGetLastError();
	//if (err != cudaSuccess) {
		//std::cout << "Kernel failed with error: " << cudaGetErrorString(err) << std::endl;
	//}
    cpy_error = hipMemcpy(vec, d_vec, vec_size*sizeof(complex), hipMemcpyDeviceToHost);
	//std::cout << "Copying to host error is: " << cpy_error << std::endl;
    hipFree(d_vec);

	
}

int main(int argc, char **argv) {

	if (argc != 7) {
		std::cout << "Input args wrong. Needs exactly six input args" << std::endl;
		exit(1);
	}


	int quid0 = atoi(argv[2]);
	int quid1 = atoi(argv[3]);
	int quid2 = atoi(argv[4]);
	int quid3 = atoi(argv[5]);
	int quid4 = atoi(argv[6]);

    //Read state vector
    std::vector<complex> state_vec;
    std::ifstream fin;
    complex temp;
	std::complex<float> std_complex_temp;


	/*
    fin.open(FILENAME);
    while(fin >> std_complex_temp) {
		temp = C(std_complex_temp.real(), std_complex_temp.imag());
        state_vec.push_back(temp);
    }
    fin.close();
	*/


    for (unsigned long i = 0; i < 1 << atoi(argv[1]); i++){ 
        //Note: normalization ignored for now
        float real = ((float) rand() / (float) (RAND_MAX));
        float imag = ((float) rand() / (float) (RAND_MAX));
        complex val = C(real, imag);
		state_vec.push_back(val);
    }


    unsigned long state_vec_size = state_vec.size();


    std::vector<complex> source_matrix_vec;
	std::cout << "here is the source matrix: " << std::endl;
    fin.open(MAT_FILENAME);
    while(fin >> std_complex_temp) {
		temp = C(std_complex_temp.real(), std_complex_temp.imag());
		std::cout << temp << std::endl;
		source_matrix_vec.push_back(temp);
    }
	if (fin.rdstate() == std::ios_base::failbit) {
		std::cout << "Ifstream failed with failbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::eofbit) {
		std::cout << "Ifstream failed with eofbit" << std::endl;
	}
	else if (fin.rdstate() == std::ios_base::badbit) {
		std::cout << "Ifstream failed with badbit" << std::endl;
	}
    fin.close();

    //Apply gate
    //Fill operator matrix in const mem
    // cudaError_t symbol_error = cudaMemcpyToSymbol(operator_matrix, source_matrix_vec.data(), MAT_DIM * MAT_DIM * sizeof(complex), 0, cudaMemcpyHostToDevice);
	// std::cout << "Error from symbol copy is: " << symbol_error << std::endl;

    run_kernel(state_vec.data(), state_vec_size, quid0, quid1, quid2, quid3, quid4, source_matrix_vec.data());


	auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
	std::cout << "GPU kernel execution time: " << duration.count() << std::endl;

	std::ofstream f_time;
	f_time.open("time_comparison.txt", std::ios_base::app);
	f_time << "GPU time: " << duration.count() << "\n";
	f_time.close();
 
/*
    std::ofstream f;
    f.open("output.txt");
    for (unsigned long i = 0; i < state_vec_size; ++i) {
        complex val = state_vec[i];
        f << val;	
    }
    f.close();
*/
    
    //debug
    std::cout << "size: " << state_vec.size() << std::endl;
    std::cout << state_vec.back() << std::endl;


    return 0;
}
