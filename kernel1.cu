//TODO: I got this off of stackoverflow. I don't know if we actually have thrust
#include <hip/hip_complex.h>
#include <complex>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <stdexcept>
#include <fstream>
#include <chrono>

#define CUDACHECK(cmd) \
    hipError_t error=cmd; \
    if(error != hipSuccess) { \
        printf("%s in %s at line %d\n", hipGetErrorString(error),__ \
        FILE__,__LINE__); \
        exit(EXIT_FAILURE); \
    }

#define FILENAME "state_vec.txt"
#define MAT_FILENAME "source_matrix.txt"
#define MAT_DIM 2
#define C(r, i) make_hipComplex(r, i)
typedef hipComplex complex;

//Overload complex number functions
__device__ __host__ complex operator*(complex a, complex b) {return hipCmulf(a,b);}
__device__ __host__ complex operator+(complex a, complex b) {return hipCaddf(a,b);}
__device__ __host__ complex operator/(complex a, complex b) {return hipCdivf(a,b);}
__device__ __host__ bool operator==(complex a, complex b) {
return a == b;
}
__device__ __host__ bool operator!=(complex a, complex b) {
return a != b;
}
__host__ std::ostream & operator << (std::ostream &out, const complex &c) {
out << "(" << hipCrealf(c);
out << ",";
out << hipCimagf(c) << ")\n";
return out;
}
__host__ std::istream & operator >> (std::istream &in, complex &c) {
    char _; //throw out variable
    double real;
    double imag;
    in >> _; //"("
    in >> real;
    in >> _; //","
    in >> imag;
    in >> _ >> _; //")\n";
    c = C(real, imag);
    return in;
}

__constant__ complex operator_matrix[MAT_DIM][MAT_DIM];

std::chrono::high_resolution_clock::time_point start, stop;

__global__ void one_qubit_kernel(complex* vec, int vec_size, int qubit_id, int elements_per_chunk) {

    //qid0 is smaller than qid
    int elements_per_thread = MAT_DIM; //1 quibit kernel
    int qid0 = qubit_id;
    int blocks_in_state_vector = ceil(vec_size / (float) (elements_per_thread * blockDim.x));
    int batch0_stride = (1 << (qid0 + 1));

    for(int global_block_id = blockIdx.x; global_block_id < blocks_in_state_vector; global_block_id += gridDim.x) {


        int element_id_base = 0;

		int batch0_id = (threadIdx.x + global_block_id * blockDim.x) / (batch0_stride/2);
		element_id_base += (threadIdx.x + global_block_id * blockDim.x) % (batch0_stride/2);
		element_id_base += batch0_id * batch0_stride;

        //iteration dependent

        complex result[MAT_DIM];
        for(int row = 0; row < MAT_DIM; row++) {
            result[row] = C(0.0f, 0.0f);
        }
        for(int i = 0; i < 2; i++) {
            int offset = (i * (1 << qid0));
            int element_id = element_id_base + offset;

            //load
            complex val;
            if(element_id < vec_size) {
                val = vec[element_id];
            }
            else {
                val = C(0.0f,0.0f);
            }

            //compute
            int column = i;
            for(int row = 0; row < MAT_DIM; row++) {
                result[row] = result[row] + (operator_matrix[row][column]*val);
            }
        }

        for(int i = 0; i < 2; i++) {
            int offset = (i * (1 << qid0));
            int element_id = element_id_base + offset;

            //store
            int row = i;
            if(element_id < vec_size) {
                vec[element_id] = result[row];
            }
        }


    }
}

//TODO: Header
template <class M>
void run_kernel(complex* vec, int vec_size, int qubit_id, M source_matrix) {
    hipDeviceSynchronize();

    //Get smem size
    hipDeviceProp_t deviceProp;
    int dev_id = 0;
    hipGetDeviceProperties(&deviceProp, dev_id);
    int smem_size_in_bytes = (int) deviceProp.sharedMemPerBlock;
    int smem_size_in_elems = smem_size_in_bytes/(2*sizeof(double));

    int max_threads_per_block = (int) deviceProp.maxThreadsPerBlock;

    // calculate
    int num_loads_to_sm = 0;
    int num_large_qubits = (1 << (qubit_id + 1)) > max_threads_per_block;
    num_loads_to_sm += num_large_qubits;
    num_loads_to_sm = (1UL << num_loads_to_sm);

//A chunk can't be larger than shared memory because we need to hold it all at once
//A chunk can't be larger than the threads in a block because we need one thread to handle each element
//A chunk can't be larger than a batch by definition
    int chunk_size = max_threads_per_block;
    int chunk_size_in_bytes = chunk_size * sizeof(complex);
    dim3 blockDim(chunk_size);
int max_grid_size = deviceProp.maxGridSize[0];
    dim3 gridDim(std::min(max_grid_size, (int) ceil(vec_size/(float)chunk_size/(float)num_loads_to_sm)));

    //print some stats about the GPU
    std::cout << "smem_size_in_elems: " << smem_size_in_elems << std::endl;
    std::cout << "max_threads_per_block: " << max_threads_per_block << std::endl;;
    std::cout << "chunk size: " << chunk_size << std::endl;;
    std::cout << "max grid size: " << max_grid_size << std::endl;;

std::cout << "Vec size (num vectors is log2): " << vec_size << std::endl;
std::cout << "kth qubit: " << qubit_id << std::endl;

    std::cout << "block dim: " << blockDim.x << std::endl;
    std::cout << "grid dim: " << gridDim.x << std::endl;

    //memcpy and run the kernel
start = std::chrono::high_resolution_clock::now();

    complex *d_vec;
    hipMalloc((void **) &d_vec, vec_size*sizeof(complex));
    hipMemcpy(d_vec, vec, vec_size*sizeof(complex), hipMemcpyHostToDevice);
    one_qubit_kernel<<<gridDim, blockDim, chunk_size_in_bytes>>>(d_vec, vec_size, qubit_id, chunk_size);
    hipDeviceSynchronize();
    hipMemcpy(vec, d_vec, vec_size*sizeof(complex), hipMemcpyDeviceToHost);
    hipFree(d_vec);

stop = std::chrono::high_resolution_clock::now();
}

int main(int argc, char **argv) {

if (argc != 3) {
std::cout << "Input args wrong. Needs total number qubits and kth qubit" << std::endl;
exit(1);
}


int kth_qubit = atoi(argv[2]);
    //int kth_qubit = 11;

    //Read state vector
    std::vector<complex> state_vec;
    std::ifstream fin;
    complex temp;
std::complex<float> std_complex_temp;
    fin.open(FILENAME);
    while(fin >> std_complex_temp) {
temp = C(std_complex_temp.real(), std_complex_temp.imag());
        state_vec.push_back(temp);
    }
if (fin.rdstate() == std::ios_base::failbit) {
std::cout << "Ifstream failed with failbit" << std::endl;
}
else if (fin.rdstate() == std::ios_base::eofbit) {
std::cout << "Ifstream failed with eofbit" << std::endl;
}
else if (fin.rdstate() == std::ios_base::badbit) {
std::cout << "Ifstream failed with badbit" << std::endl;
}
std::cout << "Vector size: " << state_vec.size() << std::endl;
    fin.close();

/*
    for (unsigned long i = 0; i < 1 << atoi(argv[1]); i++){
        //Note: normalization ignored for now
        float real = ((float) rand() / (float) (RAND_MAX));
        float imag = ((float) rand() / (float) (RAND_MAX));
        complex val = C(real, imag);
state_vec.push_back(val);
    }
*/

    unsigned long state_vec_size = state_vec.size();


//Read in source matrix
    std::vector<complex> source_matrix_vec;
std::cout << "here is the source matrix: " << std::endl;
    fin.open(MAT_FILENAME);
    while(fin >> std_complex_temp) {
temp = C(std_complex_temp.real(), std_complex_temp.imag());
        source_matrix_vec.push_back(temp);
std::cout << temp << std::endl;
    }
if (fin.rdstate() == std::ios_base::failbit) {
std::cout << "Ifstream failed with failbit" << std::endl;
}
else if (fin.rdstate() == std::ios_base::eofbit) {
std::cout << "Ifstream failed with eofbit" << std::endl;
}
else if (fin.rdstate() == std::ios_base::badbit) {
std::cout << "Ifstream failed with badbit" << std::endl;
}
    fin.close();
    //Apply gate
    //Fill operator matrix in const mem
    hipMemcpyToSymbol(HIP_SYMBOL(operator_matrix), source_matrix_vec.data(), MAT_DIM * MAT_DIM * sizeof(complex), 0, hipMemcpyHostToDevice);

    run_kernel(state_vec.data(), state_vec_size, kth_qubit, source_matrix_vec.data());


auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
std::cout << "GPU kernel execution time: " << duration.count() << std::endl;

std::ofstream f_time;
f_time.open("time_comparison.txt", std::ios_base::app);
f_time << "GPU time: " << duration.count() << "\n";
f_time.close();
 
    std::ofstream f;
    f.open("output.txt");
    for (unsigned long i = 0; i < state_vec_size; ++i) {
        complex val = state_vec[i];
        f << val;
    }
    f.close();
   
    //debug
    std::cout << "size: " << state_vec.size() << std::endl;
    std::cout << state_vec.back() << std::endl;


    return 0;
}
